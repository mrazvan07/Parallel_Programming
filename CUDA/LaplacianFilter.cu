
#include <hip/hip_runtime.h>
__global__ void LaplacianFilterKernel(unsigned char *d_img, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = x + y * width;

    if(x > 0 && x < width-1 && y > 0 && y < height-1)
    {
        int laplacian = 4*d_img[index] - d_img[index-1] - d_img[index+1] - d_img[index-width] - d_img[index+width];
        d_img[index] = min(255, max(0, laplacian));
    }
}

void LaplacianFilter(unsigned char *h_img, int width, int height)
{
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    unsigned char *d_img;
    hipMalloc((void **)&d_img, width*height*sizeof(unsigned char));
    hipMemcpy(d_img, h_img, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);

    LaplacianFilterKernel<<<grid, block>>>(d_img, width, height);

    hipMemcpy(h_img, d_img, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipFree(d_img);
}